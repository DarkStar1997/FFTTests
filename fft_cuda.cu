#include <fstream>
#include <fmt/core.h>
#include <chrono>
#include <hipfft/hipfft.h>

int main()
{
    std::ifstream in{"test_data"};
    size_t n; in >> n;
    const size_t buffer_size = 1024 * 1024;
    size_t count = 0, num = 0;
    std::vector<char> buffer; buffer.reserve(buffer_size + 100);
    hipfftComplex* d_signal;
    hipMallocManaged((void**)&d_signal, n * sizeof(hipfftComplex));
    fmt::print("Reading {} integers\n", n);

    auto file_read_start = std::chrono::steady_clock::now();
    while(count < n) {
        in.read(buffer.data(), buffer_size);
        size_t len = in.gcount();
        if(len == 0)
            break;

        bool numHasValue = false;
        for(size_t i = 0; i < len; i++) {
            const char &ch = buffer[i];
            if(ch >= '0' && ch <= '9') {
                num = num * 10 + ch - '0';
                numHasValue = true;
            }
            else if (numHasValue) {
                d_signal[count].x = num;
                d_signal[count].y = 0.0;
                num = 0;
                count++;
                numHasValue = false;
            }
        }
    }
    
    buffer.clear(); buffer.shrink_to_fit();
    in.close();
    
    auto file_read_end = std::chrono::steady_clock::now();
    fmt::println("Time taken to read from file: {}ms", std::chrono::duration_cast<std::chrono::milliseconds>(file_read_end - file_read_start).count());
    
    auto fftw_plan_start = std::chrono::steady_clock::now();
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    auto fftw_plan_end = std::chrono::steady_clock::now();
    fmt::println("Time taken to create FFT plan: {}ms", std::chrono::duration_cast<std::chrono::milliseconds>(fftw_plan_end - fftw_plan_start).count());
    
    auto fft_start = std::chrono::steady_clock::now();
    hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    auto fft_end = std::chrono::steady_clock::now();
    fmt::println("Time taken to execute FFT: {}ms", std::chrono::duration_cast<std::chrono::milliseconds>(fft_end - fft_start).count());

    auto file_write_start = std::chrono::steady_clock::now();
    std::ofstream out{"cuda_output"};
    std::string output_buffer;
    output_buffer += fmt::format("{}\n", n);
    for (size_t count = 0; count < n; ++count) {
        output_buffer += fmt::format("{}{:+f}i\n", d_signal[count].x, d_signal[count].y);
        if(output_buffer.length() >= buffer_size) {
            out.write(output_buffer.data(), output_buffer.length());
            output_buffer.clear();
        }
    }
    if(output_buffer.length() > 0) {
        out.write(output_buffer.data(), output_buffer.length());
    }
    auto file_write_end = std::chrono::steady_clock::now();
    out.close();
    fmt::println("Time taken to write to file: {}ms", std::chrono::duration_cast<std::chrono::milliseconds>(file_write_end - file_write_start).count());
    hipfftDestroy(plan);
    hipFree(d_signal);
}
